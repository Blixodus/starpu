#include "hip/hip_runtime.h"
#include <starpu.h>
#include "axpy_partition_gpu.h"
#include <stdio.h>

//This code demonstrates how to transform a kernel to execute on a given set of GPU SMs.


// Original kernel
__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n)  y[i] = a*x[i] + y[i];
}




// Transformed kernel
__global__ void saxpy_partitioned(__P_KARGS, int n, float a, float *x, float *y)
{
  __P_BEGIN;
  __P_LOOPX;
        int i = blockid.x*blockDim.x + threadIdx.x; // note that blockIdx is replaced.
	if (i<n)  y[i] = a*x[i] + y[i];
  __P_LOOPEND;
}
      

extern "C" void cuda_axpy(void *descr[], void *_args)
{
	 float a = *((float *)_args);

        unsigned n = STARPU_VECTOR_GET_NX(descr[0]);

        float *x = (float *)STARPU_VECTOR_GET_PTR(descr[0]);
        float *y = (float *)STARPU_VECTOR_GET_PTR(descr[1]);

	int SM_mapping_start = -1;
	int SM_mapping_end = -1; 
  	int SM_allocation = -1;
  
	hipStream_t stream = starpu_cuda_get_local_stream();
	int workerid = starpu_worker_get_id();
    	starpu_sched_ctx_get_sms_interval(workerid, &SM_mapping_start, &SM_mapping_end);
	SM_allocation = SM_mapping_end - SM_mapping_start;
	int dimensions = 512;	
	//partitioning setup
//	int SM_mapping_start = 0;
//  	int SM_allocation = 13;
  
	__P_HOSTSETUP(saxpy_partitioned,dim3(dimensions,1,1),dimensions,0,SM_mapping_start,SM_allocation,stream);

  	saxpy_partitioned<<<width,dimensions,0,stream>>>(__P_HKARGS,n,a,x,y);
}