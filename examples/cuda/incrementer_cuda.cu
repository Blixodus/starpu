#include "hip/hip_runtime.h"
#include "incrementer_cuda.h"

extern "C" __global__ 
void cuda_incrementer(float * tab, uint32_t nx, uint32_t pad1, float *unity, uint32_t nx2, uint32_t pad2)
{
	tab[0] = tab[0] + unity[0];
	tab[1] = tab[1] + unity[1];
	tab[2] = tab[2] + unity[2];
	
	return;
}
