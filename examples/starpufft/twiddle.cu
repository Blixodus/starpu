/*
 * StarPU
 * Copyright (C) INRIA 2009 (see AUTHORS file)
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

/* Note: these assume that the sizes are powers of two */

extern "C" __global__ void starpufftf_cuda_1d_twiddle(hipComplex * out, hipComplex * roots, unsigned n, unsigned i)
{
	unsigned j;
	unsigned start = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned end = start + 1;

	for (j = start; j < end; j++)
		out[j] = hipCmulf(out[j], roots[i*j]);
	return;
}

extern "C" void starpufftf_cuda_1d_twiddle_host(hipComplex *out, hipComplex *roots, unsigned n, unsigned i)
{
	unsigned threads_per_block = 128;

	if (n < threads_per_block) {
		dim3 dimGrid(n);
		starpufftf_cuda_1d_twiddle <<<dimGrid, 1>>> (out, roots, n, i);
	} else {
		dim3 dimGrid(n / threads_per_block);
		dim3 dimBlock(threads_per_block);
		starpufftf_cuda_1d_twiddle <<<dimGrid, dimBlock>>> (out, roots, n, i);
	}
}

extern "C" __global__ void starpufftf_cuda_2d_twiddle(hipComplex * out, hipComplex * roots0, hipComplex * roots1, unsigned n2, unsigned m2, unsigned i, unsigned j)
{
	unsigned k, l;
	unsigned startx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned starty = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned endx = startx + 1;
	unsigned endy = starty + 1;

	//for (k = startx; k < endx ; k++)
		//for (l = starty; l < endy ; l++)
	k = startx;
	l = starty;
			out[k*m2 + l] = hipCmulf(hipCmulf(out[k*m2 + l], roots0[i*k]), roots1[j*l]);
	return;
}

extern "C" void starpufftf_cuda_2d_twiddle_host(hipComplex *out, hipComplex *roots0, hipComplex *roots1, unsigned n2, unsigned m2, unsigned i, unsigned j)
{
	unsigned threads_per_dim = 16;
	if (n2 < threads_per_dim) {
		if (m2 < threads_per_dim) {
			dim3 dimGrid(n2, m2);
			starpufftf_cuda_2d_twiddle <<<dimGrid, 1>>> (out, roots0, roots1, n2, m2, i, j);
		} else {
			dim3 dimGrid(n2, threads_per_dim);
			dim3 dimBlock(1, m2 / threads_per_dim);
			starpufftf_cuda_2d_twiddle <<<dimGrid, dimBlock>>> (out, roots0, roots1, n2, m2, i, j);
		}
	} else { 
		if (m2 < threads_per_dim) {
			dim3 dimGrid(threads_per_dim, m2);
			dim3 dimBlock(n2 / threads_per_dim, 1);
			starpufftf_cuda_2d_twiddle <<<dimGrid, dimBlock>>> (out, roots0, roots1, n2, m2, i, j);
		} else {
			dim3 dimGrid(threads_per_dim, threads_per_dim);
			dim3 dimBlock(n2 / threads_per_dim, m2 / threads_per_dim);
			starpufftf_cuda_2d_twiddle <<<dimGrid, dimBlock>>> (out, roots0, roots1, n2, m2, i, j);
		}
	}
}
