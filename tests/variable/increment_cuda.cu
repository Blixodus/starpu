#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2022  Université de Bordeaux, CNRS (LaBRI UMR 5800), Inria
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void _increment_cuda_codelet(unsigned *val)
{
	val[0]++;
}

extern "C" void increment_cuda(void *descr[], void *cl_arg)
{
	unsigned *val = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[0]);

	_increment_cuda_codelet<<<1,1, 0, starpu_cuda_get_local_stream()>>>(val);
	hipError_t status = hipGetLastError();
	if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
}

extern "C" void redux_cuda_kernel(void *descr[], void *arg)
{
	(void)arg;

	STARPU_SKIP_IF_VALGRIND;

	unsigned *dst = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[0]);
	unsigned *src = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[1]);

	unsigned host_dst, host_src;

	/* This is a dummy technique of course */
	hipMemcpyAsync(&host_src, src, sizeof(unsigned), hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	hipMemcpyAsync(&host_dst, dst, sizeof(unsigned), hipMemcpyDeviceToHost, starpu_cuda_get_local_stream());
	hipStreamSynchronize(starpu_cuda_get_local_stream());

	host_dst += host_src;

	hipMemcpyAsync(dst, &host_dst, sizeof(unsigned), hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
}

extern "C" void neutral_cuda_kernel(void *descr[], void *arg)
{
	(void)arg;

	STARPU_SKIP_IF_VALGRIND;

	unsigned *dst = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[0]);

	/* This is a dummy technique of course */
	unsigned host_dst = 0;
	hipMemcpyAsync(dst, &host_dst, sizeof(unsigned), hipMemcpyHostToDevice, starpu_cuda_get_local_stream());
}
