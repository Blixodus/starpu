#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <starpu.h>

// __device__ inline double cndGPU(double d)
// {
//   const double A1 = 0.31938153f;
//   const double A2 = -0.356563782f;
//   const double A3 = 1.781477937f;
//   const double A4 = -1.821255978f;
//   const double A5 = 1.330274429f;
//   const float RSQRT2PI = 0.39894228040143267793994605993438f;

    
//   double K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

    
//   double cnd = RSQRT2PI * __expf(- 0.5f * d * d) * (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

//     if (d > 0)
//       cnd = 1.0f - cnd;

//     return cnd;
// }

__device__ inline double cndGPU(double d)
{
  return (1.0 + erf(d/sqrt(2.0)))/2.0;
}

__global__ void gpuBlackScholesKernel(double *S, double *K, double *R, double *T, 
				      double *SIG, double *CRES, double *PRES,
				      uint32_t nxS)
{
  uint32_t i, id;
  
  id = blockIdx.x * blockDim.x + threadIdx.x;
  i = id % nxS;
  
  double sqrtT = __fdividef(1.0F, rsqrtf(T[i]));
  double d1 = (log(S[i] / K[i]) + (R[i] + SIG[i] * SIG[i] * 0.5) * T[i]) / (SIG[i] * sqrt(T[i]));  
  double d2 = (log(S[i] / K[i]) + (R[i] - SIG[i] * SIG[i] * 0.5) * T[i]) / (SIG[i] * sqrt(T[i]));
  
  CRES[i] = S[i] * (normcdf(d1)) - K[i] * exp(-R[i] * T[i]) * normcdf(d2);
  PRES[i] = -S[i] * (normcdf(-d1)) + K[i] * exp(-R[i] * T[i]) * normcdf(-d2);
}

#define THREADS_PER_BLOCK 64

extern "C" void gpu_black_scholes(void *descr[], void *args)
{
  double *S, *K, *R, *T, *SIG, *CRES, *PRES;
  uint32_t nxS;
  uint32_t nblocks;

  S = (double *) STARPU_MATRIX_GET_PTR(descr[0]);
  K = (double *) STARPU_MATRIX_GET_PTR(descr[1]);
  R = (double *) STARPU_MATRIX_GET_PTR(descr[2]);
  T = (double *) STARPU_MATRIX_GET_PTR(descr[3]);
  SIG = (double *) STARPU_MATRIX_GET_PTR(descr[4]);
  CRES = (double *) STARPU_MATRIX_GET_PTR(descr[5]);
  PRES = (double *) STARPU_MATRIX_GET_PTR(descr[6]);

  nxS = STARPU_MATRIX_GET_NX(descr[0]);

  nblocks = (nxS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  gpuBlackScholesKernel
    <<< nblocks, THREADS_PER_BLOCK, 0, starpu_cuda_get_local_stream()
    >>> (S, K, R, T, SIG, CRES, PRES, nxS);
  
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}