#include "hip/hip_runtime.h"
#include "incrementer_cuda.h"

extern "C" __global__ 
void cuda_incrementer(uint32_t ptr, uint32_t nx, uint32_t ny, uint32_t ld, 
			uint32_t ptr2, uint32_t nx2, uint32_t ny2, uint32_t ld2)
{
	float *tab;
	float *unity;

	tab = (float *)ptr;
	unity = (float *)ptr2;

	tab[0] = tab[0] + unity[0];
	tab[1] = tab[1] + unity[1];
	tab[2] = tab[2] + unity[2];

	return;
}
