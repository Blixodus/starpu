//#include "spmv_cuda.h"

#include <hip/hip_runtime.h>
#include <stdint.h>

#define MIN(a,b)	((a)<(b)?(a):(b))

extern "C" __global__ 
void spmv_kernel(uint32_t nnz, uint32_t nrow, uint32_t _nzval, uint32_t _colind, uint32_t _rowptr, 
			uint32_t firstentry, uint32_t elemsize, 
			uint32_t ptr_in, uint32_t nx_in, uint32_t ny_in, uint32_t ld_in,
			uint32_t ptr_out, uint32_t nx_out, uint32_t ny_out, uint32_t ld_out)
{
	float *vecin = (float *)ptr_in;
	float *vecout = (float *)ptr_out;
	float *nzval = (float *)_nzval;

	uint32_t *rowptr = (uint32_t *)_rowptr;
	uint32_t *colind = (uint32_t *)_colind;

	/* only one dimension is used here */
	unsigned nthreads = gridDim.x*blockDim.x;
	unsigned threadid = threadIdx.x + blockIdx.x*blockDim.x;

	unsigned rowstart = threadid * ((nrow + (nthreads - 1))/nthreads);
	unsigned rowend = MIN(nrow, (threadid+1) * ((nrow + (nthreads - 1))/nthreads));

	unsigned row;
	for (row = rowstart; row < rowend; row++)
	{
		float tmp = 0.0f;
		unsigned index;

		unsigned firstindex = rowptr[row];
		unsigned lastindex = (row < nrow - 1)?rowptr[row+1]:nnz;

		for (index = firstindex; index < lastindex; index++)
		{
			unsigned col;

			col = colind[index];
			tmp += nzval[index]*vecin[col];
		}

		vecout[row] = tmp;
	}
}

extern "C" __global__ 
void spmv_kernel_2(uint32_t nnz, uint32_t nrow, uint32_t _nzval, uint32_t _colind, uint32_t _rowptr, 
			uint32_t firstentry, uint32_t elemsize, 
			uint32_t ptr_in, uint32_t nx_in, uint32_t ny_in, uint32_t ld_in,
			uint32_t ptr_out, uint32_t nx_out, uint32_t ny_out, uint32_t ld_out)
{
	float *vecin = (float *)ptr_in;
	float *vecout = (float *)ptr_out;
	float *nzval = (float *)_nzval;

	uint32_t *rowptr = (uint32_t *)_rowptr;
	uint32_t *colind = (uint32_t *)_colind;

	/* only one dimension is used here */
	unsigned block_rowstart = blockIdx.x*( (nrow + gridDim.x - 1)/gridDim.x );
	unsigned block_rowend = MIN((blockIdx.x+1)*( (nrow + gridDim.x - 1)/gridDim.x ), nrow);

	unsigned row;
	for (row = block_rowstart + threadIdx.x; row < block_rowend; row+=blockDim.x)
	{
		float tmp = 0.0f;
		unsigned index;

		unsigned firstindex = rowptr[row];
		unsigned lastindex = (row < nrow - 1)?rowptr[row+1]:nnz;

		for (index = firstindex; index < lastindex; index++)
		{
			unsigned col;

			col = colind[index];
			tmp += nzval[index]*vecin[col];
		}

		vecout[row] = tmp;
	}
	

}


extern "C" __global__ 
void spmv_kernel_3(uint32_t nnz, uint32_t nrow, uint32_t _nzval, uint32_t _colind, uint32_t _rowptr, 
			uint32_t firstentry, uint32_t elemsize, 
			uint32_t ptr_in, uint32_t nx_in, uint32_t ny_in, uint32_t ld_in,
			uint32_t ptr_out, uint32_t nx_out, uint32_t ny_out, uint32_t ld_out)
{
	float *vecin = (float *)ptr_in;
	float *vecout = (float *)ptr_out;
	float *nzval = (float *)_nzval;

	uint32_t *rowptr = (uint32_t *)_rowptr;
	uint32_t *colind = (uint32_t *)_colind;

	/* only one dimension is used here */
	unsigned block_rowstart = blockIdx.x*( (nrow + gridDim.x - 1)/gridDim.x );
	unsigned block_rowend = MIN((blockIdx.x+1)*( (nrow + gridDim.x - 1)/gridDim.x ), nrow);

	unsigned row;
	for (row = block_rowstart + threadIdx.x; row < block_rowend; row+=blockDim.x)
	{
		float tmp = 0.0f;
		unsigned index;

		unsigned firstindex = rowptr[row];
		unsigned lastindex = (row < nrow - 1)?rowptr[row+1]:nnz;

		for (index = firstindex; index < lastindex; index++)
		{
			unsigned col;

			col = colind[index];
			tmp += nzval[index]*vecin[col];
		}

		vecout[row] = tmp;
	}
	

}
