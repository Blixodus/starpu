#include "hip/hip_runtime.h"
#include <stdint.h>
#include "param.h"

#define UPDIV(a,b)	(((a) + (b) - 1) / (b))
#define MIN(a,b) ((a)<(b)?(a):(b))

__shared__ int buffer[BUFFERSIZE];

extern "C" __global__ 
#ifdef DEBUG
void bandwith_test_dumb(int *src, int *dest, unsigned size , int *p)
#else
void bandwith_test_dumb(int *src, int *dest, unsigned size)
#endif
{
	int i;
	for (i = 0; i < size; i++)
	{
		dest[i] = src[i];
	}

#ifdef DEBUG
	*p = 42;
#endif

	return;
}

extern "C" __global__ 
#ifdef DEBUG
void bandwith_test(int *src, int *dest, unsigned size , int *p)
#else
void bandwith_test(int *src, int *dest, unsigned size)
#endif
{
	unsigned i,j, actual_buffersize;
	for (i = 0; i < size ; i += BUFFERSIZE) 
	{
		__syncthreads();
		/* fill the buffer */
		actual_buffersize = MIN(BUFFERSIZE, size - i);
		for (j = 0; j < actual_buffersize; j++)
		{
			buffer[j] = src[i+j];
		}

		__syncthreads();
		/* put those data back into the global memory */
		for (j = 0; j < actual_buffersize; j++)
		{
			dest[i+j] = buffer[j];
		}

	}

#ifdef DEBUG
	*p = 42;
#endif

	return;
}


extern "C" __global__ 
#ifdef DEBUG
void bandwith_test_2(int *src, int *dest, unsigned size , int *p)
#else
void bandwith_test_2(int *src, int *dest, unsigned size)
#endif
{

	unsigned blockid = blockIdx.x + blockIdx.y*gridDim.x;
	unsigned threadid = threadIdx.x + threadIdx.y*blockDim.x;

	unsigned blockchunk_size = UPDIV(size, (gridDim.x * gridDim.y));

	unsigned blockchunk_start = MIN(blockchunk_size*blockid, size);
	unsigned blockchunk_end = MIN(blockchunk_size*(blockid+1), size);

	unsigned actual_blockchunk_size = blockchunk_end - blockchunk_start;


	unsigned threadchunk_size = UPDIV(actual_blockchunk_size, blockDim.x*blockDim.y);

	unsigned threadchunk_start = MIN(blockchunk_start + threadchunk_size*threadid, blockchunk_end);
	unsigned threadchunk_end = MIN(blockchunk_start + threadchunk_size*(threadid+1), blockchunk_end);

	unsigned i;
	for (i = threadchunk_start; i < threadchunk_end ; i++ ) 
	{
		dest[i] = src[i];
	}

#ifdef DEBUG
	*p = 42;
#endif

	return;
}

extern "C" __global__ 
#ifdef DEBUG
void bandwith_test_3(int *src, int *dest, unsigned size , int *p)
#else
void bandwith_test_3(int *src, int *dest, unsigned size)
#endif
{

	unsigned blockid = blockIdx.x + blockIdx.y*gridDim.x;
	unsigned threadid = threadIdx.x + threadIdx.y*blockDim.x;

	unsigned blockchunk_size = UPDIV(size, (gridDim.x * gridDim.y));

	unsigned blockchunk_start = MIN(blockchunk_size*blockid, size);
	unsigned blockchunk_end = MIN(blockchunk_size*(blockid+1), size);

	unsigned i;
	for (i = blockchunk_start + threadid; i < blockchunk_end ; i+=blockDim.x*blockDim.y ) 
	{
		dest[i] = src[i];
	}

#ifdef DEBUG
	*p = 42;
#endif

	return;
}
