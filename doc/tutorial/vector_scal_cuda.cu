#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010, 2011  Centre National de la Recherche Scientifique
 * Copyright (C) 2010, 2011  Université de Bordeaux 1
 *
 * Permission is granted to copy, distribute and/or modify this document
 * under the terms of the GNU Free Documentation License, Version 1.3
 * or any later version published by the Free Software Foundation;
 * with no Invariant Sections, no Front-Cover Texts, and no Back-Cover Texts.
 * See the GNU Free Documentation License in COPYING.GFDL for more details.
 */

#include <starpu.h>
#include <starpu_cuda.h>

static __global__ void vector_mult_cuda(float *val, unsigned n, float factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;
        if (i < n)
               val[i] *= factor;
}

extern "C" void scal_cuda_func(void *buffers[], void *_args)
{
        float *factor = (float *)_args;

        /* length of the vector */
        unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
        /* local copy of the vector pointer */
        float *val = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
        unsigned threads_per_block = 64;
        unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        vector_mult_cuda<<<nblocks,threads_per_block, 0, starpu_cuda_get_local_stream()>>>(val, n, *factor);

        hipStreamSynchronize(starpu_cuda_get_local_stream());
}

